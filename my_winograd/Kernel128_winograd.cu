#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>
#include <time.h>

#include "util.h"
#include "Kernel128_winograd.h"


#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}

#define MY_KERNEL 1

#define d(input, i, j, Inz) ( input[Inz + i*768 + (j<<7)] )

__global__ void kernel_128_winograd_BtdB(float *pInputs, float *pOutputs) {
	int Inx = blockIdx.x<<2, Iny0 = blockIdx.y<<2, Iny1 = threadIdx.y, Inz = threadIdx.x;
	int Iny = Iny0+Iny1, stride_r = 2048, stride_c = 128; // 2048 = 16*128
	int c_glb_start = Inx*stride_r + Iny*stride_c + Inz, c_input = Iny1*stride_c + Inz;

	extern __shared__ float input[];

	int tmp[6] = {0, 768, 1536, 2304, 3072, 3840}; // 768 = 6*128
	for (int i = 0; i < 6; i++) {
		input[c_input + tmp[i]] = pInputs[c_glb_start + i*stride_r];
	}
	__syncthreads();

	float BTd[6];
	switch(Iny1) {
		case 0:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 0, j, Inz)*4 - d(input, 2, j, Inz)*5 + d(input, 4, j, Inz);
			}
			break;
		case 1:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 + d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 2:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 - d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 3:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) + d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 4:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) - d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 5:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 3, j, Inz)*5 + d(input, 5, j, Inz);
			}
			break;
	}
	__syncthreads();

	int tmp_offset = Iny1*768+Inz;
	for (int i = 0; i < 6; i++) {
		input[tmp_offset + i*stride_c] = BTd[i];
	}
	__syncthreads();

	float BTdB[6];
	switch(Iny1) {
		case 0:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 0, Inz) - 5*d(input, i, 2, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 1:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) + d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 2:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) - d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 3:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -2*d(input, i, 1, Inz) - d(input, i, 2, Inz) + 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 4:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 2*d(input, i, 1, Inz) - d(input, i, 2, Inz) - 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 5:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 5*d(input, i, 3, Inz) + d(input, i, 5, Inz);
			}
			break;
	}
	__syncthreads();

	for (int i = 0; i < 6; i++) {
		pOutputs[(Iny1 + i*6)*2048 + (blockIdx.x*4+blockIdx.y)*128 + Inz] = BTdB[i];
	}
}


__global__ void kernel_128_OuterProduct_128(float *A, float *B, float *C) {
	int Tile = blockIdx.x, Part = blockIdx.y, tX = threadIdx.x, tY = threadIdx.y;
	int c_input = tY*128 + tX, c_kernel = c_input, T_offset = (Tile<<11) + (Part<<10) + c_input, B_offset = (Tile<<14) + c_kernel;
	
	extern __shared__ float input[];
	float *kernel = input + 1024, *out = kernel + 8192;
	int B_stride[32] = {0, 128, 256, 384, 512, 640, 768, 896, 1024, 1152, 1280, 1408, 1536, 1664, 1792, 1920, 2048, 2176, 2304, 2432, 2560, 2688, 2816, 2944, 3072, 3200, 3328, 3456, 3584, 3712, 3840, 3968};//, 4096, 4224, 4352, 4480, 4608, 4736, 4864, 4992, 5120, 5248, 5376, 5504, 5632, 5760, 5888, 6016, 6144, 6272, 6400, 6528, 6656, 6784, 6912, 7040, 7168, 7296, 7424, 7552, 7680, 7808, 7936, 8064};
	out[c_input] = 0.0f;

	input[c_input] = A[T_offset];

	for (int k = 0; k < 4; k++) {
		int B_start = B_offset + (k<<12); // 32*64
		kernel[c_kernel] = B[B_start], kernel[c_kernel+1024] = B[B_start+1024];
		kernel[c_kernel+2048] = B[B_start+2048], kernel[c_kernel+3072] = B[B_start+3072];
		__syncthreads();

		float sum = 0;
		int y_tmp = (tY<<7)+(k<<5);
		for (int j = 0; j < 32; j++) {
			sum += input[y_tmp + j] * kernel[tX + B_stride[j]];
		}
		out[tY*128 + tX] += sum;
		__syncthreads();
	}

	C[T_offset] = out[c_input];
}

int kernel_128() {
	float *input_ = get_parameter(inputName128, 16*16*128);
	float *input, *output, *l_weights;
	struct timespec start, stop;
    double runtime;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////


	/*  1. Data preparation  */
	float *t_input, *ip;
	float *kernel = get_parameter(weight_winograd_Name128, 36*128*128);

	int nInput = 16*16*128, nOutput = 16*16*128, nWeights = 36*128*128, nTransInput = 16*6*6*128, nInnerProd = 16*6*6*128;
	hipMalloc((void **) &input, nInput<<3);
	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &t_input, nTransInput<<2);
	hipMalloc((void **) &ip, nInnerProd<<2);
	hipMemset((void *) input, 0, nInput<<3);
	hipMemset((void *) output, 0, nOutput<<2);
	hipMemset((void *) t_input, 0, nTransInput<<2);
	hipMemset((void *) l_weights, 0, nWeights<<2);
	hipMemset((void *) ip, 0, nInnerProd<<2);
	hipMemcpy(input, input_, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	
	float tmp_winograd[nOutput];

	
	/*  2. Computing  */
	clock_gettime(CLOCK_REALTIME, &start);

	kernel_128_winograd_BtdB <<<dim3(4, 4), dim3(128, 6), (6*6*128)<<2 >>> (input, t_input);
	kernel_128_OuterProduct_128<<<dim3(36, 2), dim3(128, 8), (8*128 + 64*128 + 8*128)<<2 >>> (t_input, l_weights, ip);
	//cudaCheckError();
	hipDeviceSynchronize();
	
	clock_gettime(CLOCK_REALTIME, &stop);
	runtime = (stop.tv_sec - start.tv_sec) * 1e9 + (double)(stop.tv_nsec - start.tv_nsec);
	printf("Runtime = %d us\n", runtime); 


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp_winograd, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	//cudaCheckError();

	hipFree(t_input);
	hipFree(output);
	hipFree(l_weights);
	hipFree(ip);

	free(kernel);

	return runtime;
}